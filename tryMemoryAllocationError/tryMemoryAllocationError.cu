/*
 *See if we can get a GPU to throw cudaErrorMemoryAllocation
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int main(void) {
    float* ptr = NULL;
    size_t size = pow(10, 9) * sizeof(float);
    gpuErrchk( hipMalloc((void**)&ptr, size) );
    printf("Successfully allocated %zu bytes.\n", size);
    hipFree(ptr);

    /*Matrix mat = AllocateMatrix(n, k, 1);*/
    /*printMatrix(mat);*/
    /*FreeMatrix(&mat);*/

    /*cv::Mat image = cv::imread( "outputImages/result.jpg", 1 );*/
    /*printf("size = (%i, %i)\n", image.rows, image.cols);*/


    return 0;
}
